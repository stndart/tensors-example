#include "hip/hip_runtime.h"
#include "tensor.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void matrix_multiply_kernel(
    const float* A, 
    const float* B, 
    float* C, 
    int M, int N, int K
) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// CUDA matrix multiplication implementation
void cuda_matrix_multiply(const Tensor& A, const Tensor& B, Tensor& C) {
    const int M = A.rows();
    const int N = B.cols();
    const int K = A.cols();
    
    // Allocate device memory
    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
    hipError_t err;
    
    if (err = hipMalloc(&d_A, M * K * sizeof(float))) {
        throw std::runtime_error("hipMalloc d_A failed: " + std::string(hipGetErrorString(err)));
    }
    if (err = hipMalloc(&d_B, K * N * sizeof(float))) {
        hipFree(d_A);
        throw std::runtime_error("hipMalloc d_B failed: " + std::string(hipGetErrorString(err)));
    }
    if (err = hipMalloc(&d_C, M * N * sizeof(float))) {
        hipFree(d_A);
        hipFree(d_B);
        throw std::runtime_error("hipMalloc d_C failed: " + std::string(hipGetErrorString(err)));
    }
    
    // Copy data to device
    if ((err = hipMemcpy(d_A, A.data(), M * K * sizeof(float), hipMemcpyHostToDevice))) {
        goto cleanup;
    }
    if ((err = hipMemcpy(d_B, B.data(), K * N * sizeof(float), hipMemcpyHostToDevice))) {
        goto cleanup;
    }
    
    // Configure and launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (M + threadsPerBlock.y - 1) / threadsPerBlock.y
    );
    
    matrix_multiply_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    
    // Check for kernel errors
    if ((err = hipGetLastError())) {
        throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(err)));
    }
    
    // Copy result back to host
    if ((err = hipMemcpy(C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost))) {
        goto cleanup;
    }

cleanup:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    if (err != hipSuccess) {
        throw std::runtime_error("CUDA operation failed: " + std::string(hipGetErrorString(err)));
    }
}