#include "cuda_memory.h"
#include <hip/hip_runtime.h>
#include <string>

template <typename T> void cuda_allocate(T **ptr, size_t count) {
    if (err = hipMalloc(reinterpret_cast<void **>(ptr), count * sizeof(T))) {
        throw std::runtime_error("hipMalloc failed: " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <typename T> void cuda_free(T **ptr) {
    hipFree(reinterpret_cast<void **>(ptr));
}

template <typename T> void cuda_h2d(T *host, T *device, size_t count) {
    if (err = hipMemcpy(reinterpret_cast<void *>(device),
                         reinterpret_cast<void *>(host), count * sizeof(T),
                         hipMemcpyHostToDevice)) {
        throw std::runtime_error("hipMemcpy H2D failed: " +
                                 std::string(hipGetErrorString(err)));
    }
}

template <typename T> void cuda_d2h(T *host, T *device, size_t count) {
    if (err = hipMemcpy(reinterpret_cast<void *>(host),
                         reinterpret_cast<void *>(device), count * sizeof(T),
                         hipMemcpyDeviceToHost)) {
        throw std::runtime_error("hipMemcpy H2D failed: " +
                                 std::string(hipGetErrorString(err)));
    }
}
